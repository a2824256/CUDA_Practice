
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t addWithCuda(const float *index, float *delta, float *component, unsigned int size);

__global__ void initDelta(const float *index, float *res)
{
	int i = threadIdx.x;
	res[i] = 9.0 / index[i];
}

int main() {
	const int arraySize = 10;
	const float index[arraySize] = { 1.0, 2.0, 3.0, 4.0, 5.0, 6.0, 7.0, 8.0, 9.0, 10.0 };
	float delta[arraySize] = { 0 };
	float component[arraySize] = { 0 };
	hipError_t cudaStatus = addWithCuda(index, delta, component, arraySize);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	printf("{%d,%d,%d,%d,%d,%d,%d,%d,%d,%d}\n", delta[0], delta[1], delta[2], delta[3], delta[4], delta[5], delta[6], delta[7], delta[8], delta[9]);
	//for (int i = 0; i < arraySize; i++) {
		
	//}
}

//main algorithm
hipError_t addWithCuda(const float *index, float *delta, float *component, unsigned int size)
{
	float *dev_index = 0;
	float *dev_delta = 0;
	float *dev_component = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_index, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_delta, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_component, size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_index, index, size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	initDelta<<<1, size>>> (dev_index, dev_delta);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(delta, dev_delta, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_index);
	hipFree(dev_delta);
	hipFree(dev_component);

	return cudaStatus;
}
